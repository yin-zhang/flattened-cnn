#include "hip/hip_runtime.h"

#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>


/*
 * Description:
 */

__device__ int xlate_idx(int ii, int d1, int d2, int d3, int D2, int D3, int dW, int dH, int iW, int iH)
{
  int x, y, z, w;
  w = ii % d3;
  ii = ii/d3;
  z = ii % d2;
  ii = ii/d2;
  y = ii % d1;
  ii = ii/d1;
  x = ii;
  if ((w % dW != iW) || (z % dH != iH)) {
    return -1;
  }
  w = w/dW;
  z = z/dH;
  return (((x*d1+y)*D2)+z)*D3+w;
}

__device__ int xlate_idx_inv(int ii, int d1, int d2, int d3, int D2, int D3, int dW, int dH, int iW, int iH)
{
  int x, y, z, w;
  w = ii % d3;
  ii = ii/d3;
  z = ii % d2;
  ii = ii/d2;
  y = ii % d1;
  ii = ii/d1;
  x = ii;
  w = w*dW+iW;
  z = z*dH+iH;
  return (((x*d1+y)*D2)+z)*D3+w;
}

__global__ void downscale(float *input, float *output, long no_elements,
                        int dW, int dH, int iW, int iH, int d1, int d2, int d3, int D2, int D3)
{
  // output offset:
  long ii = threadIdx.x + blockDim.x * blockIdx.x;
  ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
  if (ii >= no_elements) return;
  int ipidx = xlate_idx_inv(ii, d1, d2, d3, D2, D3, dW, dH, iW, iH);
  output[ii]=input[ipidx];
}

static int cunnconv1d_SpatialSubSamplingPeriodic_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int iW = luaT_getfieldcheckint(L, 1, "iW");
  int iH = luaT_getfieldcheckint(L, 1, "iH");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  
  THCudaTensor_zero(state, output);

  input = THCudaTensor_newContiguous(state, input);
  // This is for allocating output Tensor
  long no_elements = 1;
  int i;
  for(i = 0; i < input->nDimension - 2; i++){
    no_elements *= input->size[i];
  }
  no_elements *= (input->size[i++] - iH) / dH;
  no_elements *= (input->size[i++] - iW) / dW;

  int d1;
  int d2;
  int d3;

  if (input->nDimension == 3) {
    d1 = output->size[0];
    d2 = output->size[1];
    d3 = output->size[2];
  } else {
    d1 = output->size[1];
    d2 = output->size[2];
    d3 = output->size[3];
  }

  int D2;
  int D3;

  if (input->nDimension == 3) {
    D2 = input->size[1];
    D3 = input->size[2];
  } else {
    D2 = input->size[2];
    D3 = input->size[3];
  }

  float *input_data = THCudaTensor_data(state, input);
  float *output_data = THCudaTensor_data(state, output);

  // cuda blocks & threads:
  long nthreads = 256;
  // Max number of blocks: http://en.wikipedia.org/wiki/CUDA
  // 65535 for SM 2.x, 2^32 -1 for >= 3.0
  // TODO: When we move to SM 3.5 we should update this
  long n_xblocks = min(max((int)ceil((float)no_elements / nthreads), 1), 65535);
  long n_yblocks = (long)ceil((float)no_elements / (float)(n_xblocks * nthreads));
  if (n_yblocks > 65535) {
    THError("Input size is too large!  aborting");
  }
  dim3 blocks(n_xblocks, n_yblocks);
  dim3 threads(nthreads);

  // kernel:
  downscale<<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (input_data, output_data, no_elements, dW, dH, iW, iH, d1, d2, d3, D2, D3);
  THCudaCheck(hipGetLastError());

  // final cut:
  THCudaTensor_free(state, input);

  return 1;
}

/*
 * Description:
 */
__global__ void upscale(float *gradInput_data, float *gradOutput_data, long no_elements,
                              int dW, int dH, int iW, int iH, int d1, int d2, int d3, int D2, int D3)
{
  // output offset:
  long ii = threadIdx.x + blockDim.x * blockIdx.x;
  ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
  if (ii >= no_elements) return;
  int ipidx = xlate_idx(ii, d1, d2, d3, D2, D3, dW, dH, iW, iH);
  if (ipidx >= 0) {
    gradInput_data[ii] += gradOutput_data[ipidx];
  }
}


static int cunnconv1d_SpatialSubSamplingPeriodic_updateGradInput(lua_State *L)
{

  THCState *state = getCutorchState(L);
  THCudaTensor *input =  (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int iW = luaT_getfieldcheckint(L, 1, "iW");
  int iH = luaT_getfieldcheckint(L, 1, "iH");
  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

  THCudaTensor_zero(state, gradInput);

  float *gradInput_data = THCudaTensor_data(state, gradInput);
  float *gradOutput_data = THCudaTensor_data(state, gradOutput);

  long no_elements = 1;
  for(int i = 0; i < gradInput->nDimension; i++){
    no_elements *= gradInput->size[i];
  }

  int d1;
  int d2;
  int d3;

  if (gradInput->nDimension == 3) {
    d1 = gradInput->size[0];
    d2 = gradInput->size[1];
    d3 = gradInput->size[2];
  } else {
    d1 = gradInput->size[1];
    d2 = gradInput->size[2];
    d3 = gradInput->size[3];
  }

  int D2;
  int D3;

  if (gradInput->nDimension == 3) {
    D2 = gradOutput->size[1];
    D3 = gradOutput->size[2];
  } else {
    D2 = gradOutput->size[2];
    D3 = gradOutput->size[3];
  }

  // cuda blocks & threads:
  long nthreads = 256;
  // Max number of blocks: http://en.wikipedia.org/wiki/CUDA
  // 65535 for SM 2.x, 2^32 -1 for >= 3.0
  // TODO: When we move to SM 3.5 we should update this
  long n_xblocks = min(max((int)ceil((float)no_elements / nthreads), 1), 65535);
  long n_yblocks = (long)ceil((float)no_elements / (float)(n_xblocks * nthreads));
  if (n_yblocks > 65535) {
    THError("Input size is too large!  aborting");
  }
  dim3 blocks(n_xblocks, n_yblocks);
  dim3 threads(nthreads);

  // kernel:
  upscale<<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data, no_elements,
    dW, dH, iW, iH, d1, d2, d3, D2, D3);
  THCudaCheck(hipGetLastError());

  return 1;
}


static const struct luaL_Reg cunnconv1d_SpatialSubSamplingPeriodic__ [] = {
   {"SpatialSubSamplingPeriodic_updateOutput", cunnconv1d_SpatialSubSamplingPeriodic_updateOutput},
   {"SpatialSubSamplingPeriodic_updateGradInput", cunnconv1d_SpatialSubSamplingPeriodic_updateGradInput},
   {NULL, NULL}
};


void cunnconv1d_SpatialSubSamplingPeriodic_init(lua_State *L)
{
   luaT_pushmetatable(L, "torch.CudaTensor");
   luaT_registeratname(L, cunnconv1d_SpatialSubSamplingPeriodic__, "nn");
   lua_pop(L,1);
}

