#include "hip/hip_runtime.h"

#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>


/*
 * Description:
 */

__device__ int translate_idx(int ii, int d1, int d2, int d3, int x_scale_factor, int y_scale_factor)
{
  int x, y, z, w;
  w = ii % d3;
  ii = ii/d3;
  z = ii % d2;
  ii = ii/d2;
  y = ii % d1;
  ii = ii/d1;
  x = ii;
  if ((w % y_scale_factor != 0) || (z % x_scale_factor != 0)) {
    return -1;
  }
  w = w/y_scale_factor;
  z = z/x_scale_factor;
  d2 /= y_scale_factor;
  d3 /= x_scale_factor;
  return (((x*d1+y)*d2)+z)*d3+w;
}

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int x_scale_factor, int y_scale_factor, int off_x, int off_y)
{
  int x, y, z, w;
  w = ii % d3;
  ii = ii/d3;
  z = ii % d2;
  ii = ii/d2;
  y = ii % d1;
  ii = ii/d1;
  x = ii;
  w = w*y_scale_factor+off_x;
  z = z*x_scale_factor+off_y;
  d2 *= y_scale_factor;
  d3 *= x_scale_factor;
  return (((x*d1+y)*d2)+z)*d3+w;

}

__global__ void upscale(float *input, float *output, long no_elements,
                        int x_scale_factor, int y_scale_factor, int d1, int d2, int d3)
{
  // output offset:
  long ii = threadIdx.x + blockDim.x * blockIdx.x;
  ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
  if (ii >= no_elements) return;
  int ipidx = translate_idx(ii, d1, d2, d3, x_scale_factor, y_scale_factor);
  if (ipidx >= 0) {
    output[ii]=input[ipidx];
  } else {
    output[ii] = 0;
  }
}


static int cunnconv1d_SpatialUpSamplingPeriodic_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

  int x_scale_factor = luaT_getfieldcheckint(L, 1, "x_scale_factor");
  int y_scale_factor = luaT_getfieldcheckint(L, 1, "y_scale_factor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  
  THCudaTensor_zero(state, output);

  input = THCudaTensor_newContiguous(state, input);
  // This is for allocating output Tensor
  long no_elements = 1;
  for(int i = 0; i < input->nDimension; i++){
    no_elements *= input->size[i];
  }
  no_elements *= x_scale_factor * y_scale_factor;

  int d1;
  int d2;
  int d3;

  if (input->nDimension == 3) {
    d1 = output->size[0];
    d2 = output->size[1];
    d3 = output->size[2];
  } else {
    d1 = output->size[1];
    d2 = output->size[2];
    d3 = output->size[3];
  }

  float *input_data = THCudaTensor_data(state, input);
  float *output_data = THCudaTensor_data(state, output);

  // cuda blocks & threads:
  long nthreads = 256;
  // Max number of blocks: http://en.wikipedia.org/wiki/CUDA
  // 65535 for SM 2.x, 2^32 -1 for >= 3.0
  // TODO: When we move to SM 3.5 we should update this
  long n_xblocks = min(max((int)ceil((float)no_elements / nthreads), 1), 65535);
  long n_yblocks = (long)ceil((float)no_elements / (float)(n_xblocks * nthreads));
  if (n_yblocks > 65535) {
    THError("Input size is too large!  aborting");
  }
  dim3 blocks(n_xblocks, n_yblocks);
  dim3 threads(nthreads);

  // kernel:
  upscale<<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (input_data, output_data, no_elements, x_scale_factor, y_scale_factor, d1, d2, d3);
  THCudaCheck(hipGetLastError());

  // final cut:
  THCudaTensor_free(state, input);

  return 1;
}

/*
 * Description:
 */
__global__ void downscale(float *gradInput_data, float *gradOutput_data, long no_elements,
                              int x_scale_factor, int y_scale_factor, int d1, int d2, int d3)
{
  // output offset:
  long ii = threadIdx.x + blockDim.x * blockIdx.x;
  ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
  if (ii >= no_elements) return;
  int ipidx = translate_idx_inv(ii, d1, d2, d3, x_scale_factor, y_scale_factor, 0, 0);
  gradInput_data[ii] += gradOutput_data[ipidx];
}


static int cunnconv1d_SpatialUpSamplingPeriodic_updateGradInput(lua_State *L)
{

  THCState *state = getCutorchState(L);
  THCudaTensor *input =  (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

  int x_scale_factor = luaT_getfieldcheckint(L, 1, "x_scale_factor");
  int y_scale_factor = luaT_getfieldcheckint(L, 1, "y_scale_factor");
  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

  THCudaTensor_zero(state, gradInput);

  float *gradInput_data = THCudaTensor_data(state, gradInput);
  float *gradOutput_data = THCudaTensor_data(state, gradOutput);

  long no_elements = 1;
  for(int i = 0; i < gradInput->nDimension; i++){
    no_elements *= gradInput->size[i];
  }

  int d1;
  int d2;
  int d3;

  if (gradInput->nDimension == 3) {
    d1 = gradInput->size[0];
    d2 = gradInput->size[1];
    d3 = gradInput->size[2];
  } else {
    d1 = gradInput->size[1];
    d2 = gradInput->size[2];
    d3 = gradInput->size[3];
  }

  // cuda blocks & threads:
  long nthreads = 256;
  // Max number of blocks: http://en.wikipedia.org/wiki/CUDA
  // 65535 for SM 2.x, 2^32 -1 for >= 3.0
  // TODO: When we move to SM 3.5 we should update this
  long n_xblocks = min(max((int)ceil((float)no_elements / nthreads), 1), 65535);
  long n_yblocks = (long)ceil((float)no_elements / (float)(n_xblocks * nthreads));
  if (n_yblocks > 65535) {
    THError("Input size is too large!  aborting");
  }
  dim3 blocks(n_xblocks, n_yblocks);
  dim3 threads(nthreads);

  // kernel:
  downscale<<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data, no_elements,
    x_scale_factor, y_scale_factor, d1, d2, d3);
  THCudaCheck(hipGetLastError());

  return 1;
}


static const struct luaL_Reg cunnconv1d_SpatialUpSamplingPeriodic__ [] = {
   {"SpatialUpSamplingPeriodic_updateOutput", cunnconv1d_SpatialUpSamplingPeriodic_updateOutput},
   {"SpatialUpSamplingPeriodic_updateGradInput", cunnconv1d_SpatialUpSamplingPeriodic_updateGradInput},
   {NULL, NULL}
};


void cunnconv1d_SpatialUpSamplingPeriodic_init(lua_State *L)
{
   luaT_pushmetatable(L, "torch.CudaTensor");
   luaT_registeratname(L, cunnconv1d_SpatialUpSamplingPeriodic__, "nn");
   lua_pop(L,1);
}

