#include "hip/hip_runtime.h"
#include "utils.h"
#include "common.h"

// Kernel for fast unfold+copy
// (borrowed from Caffe: https://github.com/BVLC/caffe/blob/master/src/caffe/layers/conv_layer.cu)
__global__ void im2col_kernel_h(const int n, const float* data_im,
    const int height, const int width, const int ksize_h, const int ksize_w, const int pad_h,
    const int pad_w, const int stride_h, const int stride_w, const int height_col, const int width_col,
    float* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % width_col;
    index /= width_col;
    int h_out = index % height_col;
    int channel_in = index / height_col;
    int channel_out = channel_in * ksize_h * ksize_w;
    int h_in = h_out * stride_h - pad_h;
    int w_in = w_out * stride_w - pad_w;
    data_col += (channel_out * height_col + h_out) * width_col + w_out;
    data_im += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < ksize_h; ++i) {
      for (int j = 0; j < ksize_w; ++j) {
        int h = h_in + i;
        int w = w_in + j;
        *data_col = (h >= 0 && w >= 0 && h < height && w < width) ?
          data_im[i * width + j] : 0;
        data_col += height_col * width_col;
      }
    }
  }
}


__global__ void conv_horizontal_naive_output(const int n, float *y,
                                             const float *x, const float *w,
                                             const int iH, const int iW, const int kL)
{
   for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < n; i += blockDim.x*gridDim.x) {
      int oW = iW - kL + 1;
      int x_offset = (i/oW)*iW + i%oW;
      int w_offset = (i/(oW*iH))*kL;

      for (int k = 0; k < kL; k++) {
         y[i] += w[w_offset + k]*x[x_offset + k];
      }
   }
}


__global__ void conv_horizontal_naive_gradInput(const int n, float *dx,
                                                const float *dy, const float *w,
                                                const int oH, const int oW, const int kL)
{
   for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < n; i += blockDim.x*gridDim.x) {
      int iW = oW + kL - 1;
      int col = i%iW;
      int dy_offset = (i/iW)*oW + i%iW;
      int w_offset = (i/(iW*oH))*kL;

      int k_begin = max(0, col-oW+1);
      int k_end = min(kL, col+1);

      dx[i] = 0.0f;
      for (int k = k_begin; k < k_end; k++) {
         dx[i] += w[w_offset + k]*dy[dy_offset - k];
      }
   }
}


__global__ void conv_horizontal_naive_gradParam(const int n, float *dw,
                                                const float *x, const float *dy,
                                                const int kL, const int oH, const int oW)
{
   for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < n; i += blockDim.x*gridDim.x) {
      int iW = oW + kL - 1;
      int dy_offset = (i/kL)*oH*oW;
      int x_offset = (i/kL)*oH*oW + i%kL;

      for (int j = 0; j < oH; j++) {
         for (int k = 0; k < oW; k++) {
            dw[i] += dy[dy_offset + j*oW + k]*x[x_offset + j*iW + k];
         }
      }
   }
}


__global__ void conv_horizontal_naive_gradWeight(const int n, float *y,
                                                 const float *x, const int kL,
                                                 const int iC)
{
   for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < n; i += blockDim.x*gridDim.x) {
      y[i] = x[(i/kL)*kL*iC + i];
   }
}


static int cunnconv1d_HorizontalConvolution_updateOutput(lua_State *L) {
   THCState *state = getCutorchState(L);
   THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

   int nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");
   int nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");

   THCudaTensor *weight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
   THCudaTensor *bias = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
   THCudaTensor *ones = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "ones", "torch.CudaTensor");
   THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

   const int device = THCudaTensor_getDevice(state, weight);
   luaL_argcheck(L, THCudaTensor_getDevice(state, bias) == device, 1,
                 "weight and bias need to be on the same device");
   luaL_argcheck(L, THCudaTensor_getDevice(state, output) == device ||
                 THCudaTensor_getDevice(state, output) == -1, 1,
                 "weight and output need to be on the same device");
   luaL_argcheck(L, THCudaTensor_getDevice(state, input) == device, 2,
                 "weight and input need to be on the same device");
   luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2,
                 "3D or 4D (batch mode) tensor is expected");

   // change to batch mode
   int batch = 1;
   if (input->nDimension == 3) {
      luaL_argcheck(L, input->size[0] == nInputPlane, 2, "input channels and nInputPlane dont match");
      batch = 0;
      THCudaTensor_resize4d(state, input, 1, input->size[0], input->size[1], input->size[2]);
   } else {
      luaL_argcheck(L, input->size[1] == nInputPlane, 2, "input channels and nInputPlane dont match");
   }

   long batchSize    = input->size[0];
   long inputHeight  = input->size[2];
   long inputWidth   = input->size[3];
   long outputHeight = inputHeight;
   long outputWidth  = inputWidth - weight->size[1] + 1;

   THCudaTensor_resize4d(state, output, batchSize, nOutputPlane, outputHeight, outputWidth);

   if (ones->nDimension != 2 || ones->size[0]*ones->size[1] < outputHeight*outputWidth) {
      THCudaTensor_resize2d(state, ones, outputHeight, outputWidth);
      THCudaTensor_fill(state, ones, 1);
   }

   THCudaTensor *input_n = THCudaTensor_new(state);
   THCudaTensor *output_n = THCudaTensor_new(state);

   for (int elt = 0; elt < batchSize; elt ++) {

      // select each batch
      THCudaTensor_select(state, input_n, input, 0, elt);
      THCudaTensor_select(state, output_n, output, 0, elt);

      // fill biases
      THCudaBlas_Sgemm(
         state, 't', 'n',
         outputHeight*outputWidth, nOutputPlane, 1,
         1,
         THCudaTensor_data(state, ones), 1,
         THCudaTensor_data(state, bias), 1,
         0,
         THCudaTensor_data(state, output_n), outputHeight*outputWidth
      );

      // convolve
      long num_threads = nOutputPlane*outputHeight*outputWidth;
      conv_horizontal_naive_output <<<GET_BLOCKS(num_threads), CUDA_NUM_THREADS>>>
         (num_threads,
          THCudaTensor_data(state, output_n),
          THCudaTensor_data(state, input_n),
          THCudaTensor_data(state, weight),
          inputHeight, inputWidth, weight->size[1]);
   }

   THCudaTensor_free(state, input_n);
   THCudaTensor_free(state, output_n);

   // revert to single batch
   if (batch == 0) {
      THCudaTensor_resize3d(state, output, nOutputPlane, outputHeight, outputWidth);
      THCudaTensor_resize3d(state, input, nInputPlane, inputHeight, inputWidth);
   }

   return 1;
}


static int cunnconv1d_HorizontalConvolution_updateGradInput(lua_State *L) {
   THCState *state = getCutorchState(L);
   THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
   THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

   int nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");
   int nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");

   THCudaTensor *weight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
   THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

   const int device = THCudaTensor_getDevice(state, weight);
   luaL_argcheck(L, THCudaTensor_getDevice(state, input) == device, 2,
                 "weight and input need to be on the same device");
   luaL_argcheck(L, THCudaTensor_getDevice(state, gradInput) == device
                 || THCudaTensor_getDevice(state, gradInput) == -1, 2,
                 "weight and gradInput need to be on the same device");
   luaL_argcheck(L, THCudaTensor_getDevice(state, gradOutput) == device
                 || THCudaTensor_getDevice(state, gradOutput) == -1, 2,
                 "weight and gradOutput need to be on the same device");
   luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2,
                 "3D or 4D (batch mode) tensor is expected");

   int batch = 1;
   if (input->nDimension == 3) {
      batch = 0;
      THCudaTensor_resize4d(state, input, 1, input->size[0], input->size[1], input->size[2]);
      THCudaTensor_resize4d(state, gradOutput, 1, gradOutput->size[0], gradOutput->size[1], gradOutput->size[2]);
   }

   long batchSize    = input->size[0];
   long inputHeight  = input->size[2];
   long inputWidth   = input->size[3];
   long outputHeight = inputHeight;
   long outputWidth  = inputWidth - weight->size[1] + 1;

   THCudaTensor_resize4d(state, gradInput, batchSize, nInputPlane, inputHeight, inputWidth);

   THCudaTensor *gradInput_n = THCudaTensor_new(state);
   THCudaTensor *gradOutput_n = THCudaTensor_new(state);

   for (int elt = 0; elt < batchSize; elt ++) {

      // select each batch in 2D
      THCudaTensor_select(state, gradInput_n, gradInput, 0, elt);
      THCudaTensor_select(state, gradOutput_n, gradOutput, 0, elt);

      // convolve
      long num_threads = nInputPlane*inputHeight*inputWidth;
      conv_horizontal_naive_gradInput <<<GET_BLOCKS(num_threads), CUDA_NUM_THREADS>>>
         (num_threads,
          THCudaTensor_data(state, gradInput_n),
          THCudaTensor_data(state, gradOutput_n),
          THCudaTensor_data(state, weight),
          outputHeight, outputWidth, weight->size[1]);
   }

   THCudaTensor_free(state, gradInput_n);
   THCudaTensor_free(state, gradOutput_n);

   // revert to single batch
   if (batch == 0) {
      THCudaTensor_resize3d(state, input, nInputPlane, inputHeight, inputWidth);
      THCudaTensor_resize3d(state, gradInput, nInputPlane, inputHeight, inputWidth);
      THCudaTensor_resize3d(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
   }

   return 1;
}


static int cunnconv1d_HorizontalConvolution_accGradParameters(lua_State *L) {
   THCState *state = getCutorchState(L);
   THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
   THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

   float scale = luaL_optnumber(L, 4, 1);
   int nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");
   int nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
   int kL = luaT_getfieldcheckint(L, 1, "kL");

   THCudaTensor *gradWeight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
   THCudaTensor *gradBias = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradBias", "torch.CudaTensor");
   THCudaTensor *ones = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "ones", "torch.CudaTensor");
   THCudaTensor *finput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "finput", "torch.CudaTensor");
   THCudaTensor *fgradWeight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "fgradWeight", "torch.CudaTensor");

   const int device = THCudaTensor_getDevice(state, gradWeight);
   luaL_argcheck(L, THCudaTensor_getDevice(state, gradBias) == device, 1,
                 "gradWeight and gradBias need to be on the same device");
   luaL_argcheck(L, THCudaTensor_getDevice(state, input) == device, 1,
                 "gradWeight and input need to be on the same device");
   luaL_argcheck(L, THCudaTensor_getDevice(state, gradOutput) == device, 1,
                 "gradWeight and gradOutput need to be on the same device");
   luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2,
                 "3D or 4D (batch mode) tensor is expected");

   // change to batch mode
   int batch = 1;
   if (input->nDimension == 3) {
      batch = 0;
      THCudaTensor_resize4d(state, input, 1, input->size[0], input->size[1], input->size[2]);
      THCudaTensor_resize4d(state, gradOutput, 1, gradOutput->size[0], gradOutput->size[1], gradOutput->size[2]);
   }

   long batchSize    = input->size[0];
   long inputHeight  = input->size[2];
   long inputWidth   = input->size[3];
   long outputHeight = inputHeight;
   long outputWidth  = inputWidth - kL + 1;

   if (ones->nDimension != 2 || ones->size[0]*ones->size[1] < outputHeight*outputWidth) {
      THCudaTensor_resize2d(state, ones, outputHeight, outputWidth);
      THCudaTensor_fill(state, ones, 1);
   }

   THCudaTensor_resize2d(state, finput, kL*nInputPlane, outputHeight*outputWidth);
   THCudaTensor_resize2d(state, fgradWeight, nOutputPlane, kL*nInputPlane);

   THCudaTensor *input_n = THCudaTensor_new(state);
   THCudaTensor *gradOutput_n = THCudaTensor_new(state);

   for (int elt = 0; elt < batchSize; elt ++) {

      // select each batch
      THCudaTensor_select(state, input_n, input, 0, elt);
      THCudaTensor_select(state, gradOutput_n, gradOutput, 0, elt);

      // unroll
      long num_threads = nInputPlane*outputHeight*outputWidth;
      im2col_kernel_h <<<GET_BLOCKS(num_threads), CUDA_NUM_THREADS>>> (
         num_threads,
         THCudaTensor_data(state, input_n),
         inputHeight, inputWidth, 1, kL, 0, 0, 1, 1,
         outputHeight, outputWidth,
         THCudaTensor_data(state, finput)
      );

      // convolve
      THCudaBlas_Sgemm(
         state, 't', 'n',
         kL*nInputPlane, nOutputPlane, outputHeight*outputWidth,
         scale,
         THCudaTensor_data(state, finput), outputHeight*outputWidth,
         THCudaTensor_data(state, gradOutput_n), outputHeight*outputWidth,
         (elt > 0),
         THCudaTensor_data(state, fgradWeight), kL*nInputPlane
      );

      // fill biases
      THCudaBlas_Sgemv(
         state,
         't',
         outputHeight*outputWidth, nOutputPlane,
         scale,
         THCudaTensor_data(state, gradOutput_n), outputHeight*outputWidth,
         THCudaTensor_data(state, ones), 1,
         1,
         THCudaTensor_data(state, gradBias), 1
      );
   }

   // extract gradWeight
   long num_threads_ = kL*nInputPlane;
   conv_horizontal_naive_gradWeight <<<GET_BLOCKS(num_threads_), CUDA_NUM_THREADS>>> (
      num_threads_,
      THCudaTensor_data(state, gradWeight),
      THCudaTensor_data(state, fgradWeight),
      kL, nInputPlane
   );

   THCudaTensor_free(state, input_n);
   THCudaTensor_free(state, gradOutput_n);

   if (batch == 0) {
      THCudaTensor_resize3d(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
      THCudaTensor_resize3d(state, input, nInputPlane, inputHeight, inputWidth);
   }

   return 0;
}


static const struct luaL_Reg cunnconv1d_HorizontalConvolution__ [] = {
   {"HorizontalConvolution_updateOutput", cunnconv1d_HorizontalConvolution_updateOutput},
   {"HorizontalConvolution_updateGradInput", cunnconv1d_HorizontalConvolution_updateGradInput},
   {"HorizontalConvolution_accGradParameters", cunnconv1d_HorizontalConvolution_accGradParameters},
   {NULL, NULL}
};


void cunnconv1d_HorizontalConvolution_init(lua_State *L)
{
   luaT_pushmetatable(L, "torch.CudaTensor");
   luaT_registeratname(L, cunnconv1d_HorizontalConvolution__, "nn");
   lua_pop(L,1);
}
